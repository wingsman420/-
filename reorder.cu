#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <cmath>
#include <stdlib.h>

//定义LHS矩阵大小
#define M 4 //row
#define K 4 //column
#define LHS_SIZE M*K

//定义RHS矩阵大小
#define N 1//column
#define RHS_SIZE K*N

#define RESULT_SIZE M*N

//定义子矩阵大小
#define BLOCK_ROW_SIZE 2
#define BLOCK_COL_SIZE 2

typedef struct{
    int col_id;
    int num;
}COLUMN;

int cmp(const void *a,const void *b){
    return (*(COLUMN*)b).num-(*(COLUMN*)a).num;
}

//得到行重排数组
void get_row(int row[],float h_LHS[]){
    int row_index=0;
    int i,j,k;
    int row_flag[M]={0};
    for(i=0;i<M;i++){
        for(j=0;j<K;j++){
            if(fabs(h_LHS[i*M+j])>1e-6&&row_flag[i]==0){
                row[row_index++]=i;
                row_flag[i]=1;
                for(k=i;k<M;k++){
                    if(fabs(h_LHS[k*M+j])>1e-6&&row_flag[k]==0){
                        row[row_index++]=k;
                        row_flag[k]=1;
                    }
                    if(row_index==M)
                        break;
                }
            }
            if(row_index==M)
                break;
        }
    }
}

//得到列重排数组
void get_col(COLUMN h_col[],float h_LHS[],int row[]){
    int i,j;
    for(i=0;i<K;i++){
        for(j=0;j<M/BLOCK_ROW_SIZE;j++)
            h_col[j*K+i].col_id=i;
        for(j=0;j<M;j++)
            if(fabs(h_LHS[row[j]*M+i])>1e-6)
                h_col[j/BLOCK_ROW_SIZE*K+i].num++;
    }
    for(i=0;i<M/BLOCK_ROW_SIZE;i++)
        qsort(h_col+i*K,K,sizeof(COLUMN),cmp);
    
}

//对LHS矩阵进行行重排
__global__ void LHS_row_reorder(float *out,float *in,int *row){
    int row_id=blockIdx.y*M+threadIdx.y;
    int col_id=blockIdx.x*K+threadIdx.x;
    int index_out=row_id*M+col_id;
    int index_in=row[row_id]*M+col_id;
    out[index_out]=in[index_in];
}

//对LHS矩阵进行列重排
__global__ void LHS_col_reorder(float *out,float *in,COLUMN *col){
    int row_id=blockIdx.y*M+threadIdx.y;
    int c_id=blockIdx.x*K+threadIdx.x;
    int index_out=row_id*M+c_id;
    int index_in;
    index_in=row_id*M+col[row_id/BLOCK_ROW_SIZE*K+c_id].col_id;
    out[index_out]=in[index_in];
}

__global__ void get_nz_block(int *out,float *in){
    int row_id=blockIdx.y*M/BLOCK_ROW_SIZE+threadIdx.y;
    int col_id=blockIdx.x*K/BLOCK_COL_SIZE+threadIdx.x;
    int index_out=row_id*M/BLOCK_ROW_SIZE+col_id;
    int i,j;
    out[index_out]=0;
    for(i=0;i<BLOCK_ROW_SIZE;i++)
        for(j=0;j<BLOCK_COL_SIZE;j++)
            out[index_out]+=(fabs(in[(i+row_id*2)*M+j+col_id*2])>1e-6);
}


COLUMN h_col[(M/BLOCK_ROW_SIZE)*K];//列重排数组

int main(){
    // 创建一个 cuSPARSE 句柄
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    //在主机上创建并初始化LHS矩阵
    float h_LHS[]={1,0,0,0,0,2,0,0,3,0,1,0,0,4,0,0};

    //得到行重排数组
    int h_row[M]={0};
    get_row(h_row,h_LHS);

    //得到列重排数组
    get_col(h_col,h_LHS,h_row);

    //在GPU上声明变量
    float *d_LHS,*d_temp,*d_reordered;
    int *d_row;
    COLUMN *d_col;

    //申请GPU上的内存
    hipMalloc((void**)&d_LHS,sizeof(float)*LHS_SIZE);
    hipMalloc((void**)&d_temp,sizeof(float)*LHS_SIZE);
    hipMalloc((void**)&d_reordered,sizeof(float)*LHS_SIZE);
    hipMalloc((void**)&d_row,sizeof(int)*M);
    hipMalloc((void**)&d_col,sizeof(COLUMN)*(M/BLOCK_ROW_SIZE)*K);

    //将数据复制到GPU
    hipMemcpy(d_LHS,h_LHS,sizeof(float)*LHS_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(d_row,h_row,sizeof(int)*M,hipMemcpyHostToDevice);
    hipMemcpy(d_col,h_col,sizeof(COLUMN)*(M/BLOCK_ROW_SIZE)*K,hipMemcpyHostToDevice);
    
    dim3 blockSize(M,K);
    dim3 gridSize(1,1);
    //对LHS矩阵进行行重排
    LHS_row_reorder<<<gridSize,blockSize>>>(d_temp,d_LHS,d_row);
    //对LHS矩阵进行列重排
    LHS_col_reorder<<<gridSize,blockSize>>>(d_reordered,d_temp,d_col);
    
    //得到非零块数量
    int *d_nz_block;
    hipMalloc((void**)&d_nz_block,sizeof(int)*(M/BLOCK_ROW_SIZE)*(K/BLOCK_COL_SIZE));
    dim3 subBlockSize(M/BLOCK_ROW_SIZE,K/BLOCK_COL_SIZE);
    
    get_nz_block<<<gridSize,subBlockSize>>>(d_nz_block,d_reordered);

    //得到非零块数组
    int *h_nz_block=(int*)malloc(sizeof(int)*(M/BLOCK_ROW_SIZE)*(K/BLOCK_COL_SIZE));
    hipMemcpy(h_nz_block,d_nz_block,sizeof(int)*(M/BLOCK_ROW_SIZE)*(K/BLOCK_COL_SIZE),hipMemcpyDeviceToHost);
    float *h_reordered=(float*)malloc(sizeof(float)*LHS_SIZE);
    hipMemcpy(h_reordered,d_reordered,sizeof(float)*LHS_SIZE,hipMemcpyDeviceToHost);

    int nnzb=0;
    int i,j;

    int *h_bsrRowPtr=(int*)malloc(sizeof(int)*(M/BLOCK_ROW_SIZE+1));
    h_bsrRowPtr[0]=0;
    for(i=0;i<M/BLOCK_ROW_SIZE;i++){
        for(j=0;j<K/BLOCK_COL_SIZE;j++){
            nnzb+=(h_nz_block[i*M/BLOCK_ROW_SIZE+j]>0);
        }
        h_bsrRowPtr[i+1]=nnzb;
    }


    int *h_bsrColInd=(int*)malloc(sizeof(int)*nnzb);
    int index=0;
    for(i=0;i<M/BLOCK_ROW_SIZE;i++){
        for(j=0;j<K/BLOCK_COL_SIZE;j++){
            if(h_nz_block[i*M/BLOCK_ROW_SIZE+j]>0){
                h_bsrColInd[index++]=j;
            }
        }
    }

    index=0;
    float *h_bsrVal=(float*)malloc(sizeof(float)*nnzb*BLOCK_ROW_SIZE*BLOCK_COL_SIZE);
    for(i=0;i<M;i++)
        for(j=0;j<K;j++)
            if(h_nz_block[(i/BLOCK_ROW_SIZE)*M/BLOCK_ROW_SIZE+j/BLOCK_COL_SIZE]>0)
                h_bsrVal[index++]=h_reordered[i*M+j];

    float *d_bsrVal;
    int *d_bsrRowPtr,*d_bsrColInd;

    hipMalloc((void**)&d_bsrVal,sizeof(float)*nnzb*BLOCK_ROW_SIZE*BLOCK_COL_SIZE);
    hipMalloc((void**)&d_bsrRowPtr,sizeof(int)*(M/BLOCK_ROW_SIZE+1));
    hipMalloc((void**)&d_bsrColInd,sizeof(int)*nnzb);

    hipMemcpy(d_bsrVal,h_bsrVal,sizeof(float)*nnzb*BLOCK_ROW_SIZE*BLOCK_COL_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(d_bsrRowPtr,h_bsrRowPtr,sizeof(int)*(M/BLOCK_ROW_SIZE+1),hipMemcpyHostToDevice);
    hipMemcpy(d_bsrColInd,h_bsrColInd,sizeof(int)*nnzb,hipMemcpyHostToDevice);

    //初始化RHS矩阵(K*N)
    float h_RHS[]={0,2,1,4};
    float h_RHS1[RHS_SIZE];
    float h_RHS2[RHS_SIZE];
    for(i=0;i<K;i++){
        h_RHS1[i]=h_RHS[h_col[i].col_id];
    }
    std::cout<<std::endl;
    for(i=0;i<K;i++){
        h_RHS2[i]=h_RHS[h_col[K+i].col_id];
    }
    float *d_RHS,*d_RHS1,*d_RHS2,*d_result1,*d_result2;
    hipMalloc((void**)&d_RHS,sizeof(float)*RHS_SIZE);
    hipMalloc((void**)&d_RHS1,sizeof(float)*RHS_SIZE);
    hipMalloc((void**)&d_RHS2,sizeof(float)*RHS_SIZE);
    hipMalloc((void**)&d_result1,sizeof(float)*RESULT_SIZE);
    hipMalloc((void**)&d_result2,sizeof(float)*RESULT_SIZE);
    hipMemcpy(d_RHS,h_RHS,sizeof(float)*RHS_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(d_RHS1,h_RHS1,sizeof(float)*RHS_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(d_RHS2,h_RHS2,sizeof(float)*RHS_SIZE,hipMemcpyHostToDevice);
    
    float alpha=1.0f;
    float beta=0.0f;
    hipsparseMatDescr_t descr{nullptr};
    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseSbsrmm(
        handle,
        HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        M/BLOCK_ROW_SIZE,
        N,
        K/BLOCK_COL_SIZE,
        nnzb,
        &alpha,
        descr,
        d_bsrVal,
        d_bsrRowPtr,
        d_bsrColInd,
        BLOCK_ROW_SIZE,
        d_RHS1,
        K,
        &beta,
        d_result1,
        M
    );

    hipsparseSbsrmm(
        handle,
        HIPSPARSE_DIRECTION_ROW,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        M/BLOCK_ROW_SIZE,
        N,
        K/BLOCK_COL_SIZE,
        nnzb,
        &alpha,
        descr,
        d_bsrVal,
        d_bsrRowPtr,
        d_bsrColInd,
        BLOCK_ROW_SIZE,
        d_RHS2,
        K,
        &beta,
        d_result2,
        M
    );

    float *h_result1=(float*)malloc(sizeof(float)*RESULT_SIZE);
    float *h_result2=(float*)malloc(sizeof(float)*RESULT_SIZE);
    hipMemcpy(h_result1,d_result1,sizeof(float)*RESULT_SIZE,hipMemcpyDeviceToHost);
    hipMemcpy(h_result2,d_result2,sizeof(float)*RESULT_SIZE,hipMemcpyDeviceToHost);

    float *h_result=(float*)malloc(sizeof(float)*RESULT_SIZE);
    for(i=0;i<M;i++){
        if(i<M/2){
            h_result[h_row[i]]=h_result1[i];
        }
        else{
            h_result[h_row[i]]=h_result2[i];
        }
    }
    //check
    for(i=0;i<M;i++){
        std::cout<<h_result[i]<<" ";
    }
    std::cout<<std::endl;
    //释放GPU内存
    hipsparseDestroy(handle);

    hipFree(d_LHS);
    hipFree(d_temp);
    hipFree(d_reordered);
    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_bsrColInd);
    hipFree(d_bsrRowPtr);
    hipFree(d_bsrVal);
    hipFree(d_nz_block);
    hipFree(d_RHS1);
    hipFree(d_RHS2);
    hipFree(d_RHS);
    hipFree(d_result1);
    hipFree(d_result2);

    return 0;
}
